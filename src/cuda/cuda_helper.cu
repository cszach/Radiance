#include "cuda_helper.cuh"
#include <iostream>

void checkCuda(hipError_t result, char const *const function_name,
               const char *const filename, int const line_num) {
  if (result) {
    std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at "
              << filename << ":" << line_num << " '" << function_name
              << "': " << hipGetErrorString(result) << "\n";
    hipDeviceReset();
    exit(99);
  }
}