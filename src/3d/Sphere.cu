#include "hip/hip_runtime.h"
#include "Object.cuh"
#include "Ray.cuh"
#include "Sphere.cuh"
#include "Vec3.cuh"
#include <cstdio>

__device__ Sphere::Sphere(float _radius, Material *_material)
    : radius(_radius) {
  material = _material;
}

__device__ bool Sphere::hit(const Ray &ray, Interval ray_t,
                            HitRecord &rec) const {
  Vec3 o_c = ray.origin - position; // ray origin - sphere position
  float a = ray.direction.dot(ray.direction);
  float half_b = ray.direction.dot(o_c);
  float c = o_c.dot(o_c) - radius * radius;

  float discriminant = half_b * half_b - a * c;

  // Find the nearest root that lies in the acceptable range

  float sqrt_d = sqrt(discriminant);
  float root = (-half_b - sqrt_d) / a;

  if (!ray_t.surrounds(root)) {
    root = (-half_b + sqrt_d) / a;
    if (!ray_t.surrounds(root)) {
      return false;
    }
  }

  rec.t = root;
  rec.p = ray.at(root);
  Vec3 outward_normal = (rec.p - position) / radius;
  rec.setFaceNormal(ray, outward_normal);
  rec.material = material;

  return true;
}

__device__ void Sphere::computeBoundingBox() {
  Vec3 halfExtents = Vec3(radius, radius, radius);
  boundingBox = AABB(position - halfExtents, position + halfExtents);
}
