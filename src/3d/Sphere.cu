#include "hip/hip_runtime.h"
#include "Sphere.hpp"

#include "Ray.hpp"
#include "Vec3.hpp"

__device__ Sphere::Sphere(float radius) : _radius(radius) {}

__device__ bool Sphere::hit(const Ray &ray, float t_min, float t_max,
                            HitRecord &rec) const {
  Vec3 o_c = ray.origin - getPosition(); // ray origin - sphere position
  float a = ray.direction.dot(ray.direction);
  float half_b = ray.direction.dot(o_c);
  float c = o_c.dot(o_c) - _radius * _radius;

  float discriminant = half_b * half_b - a * c;

  // Find the nearest root that lies in the acceptable range

  float sqrt_d = sqrt(discriminant);
  float root = (-half_b - sqrt_d) / a;

  if (root < t_min || root > t_max) {
    root = (-half_b + sqrt_d) / a;
    if (root < t_min || root > t_max) {
      return false;
    }
  }

  rec.t = root;
  rec.p = ray.at(root);
  Vec3 outward_normal = (rec.p - getPosition()) / _radius;
  rec.setFaceNormal(ray, outward_normal);

  return true;
}
