#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
// #include <iostream>

#include "MathUtils.cuh"
#include "Renderer.cuh"
#include "cuda_helper.cuh"

__global__ void p_render(Scene **scene, DRenderer **renderer, float *fb,
                         hiprandState *rand_state);
__global__ void setupRenderer(DRenderer **d_renderer, Camera **d_camera,
                              int output_width, int output_height);

__host__ Renderer::Renderer(Camera **d_camera, int output_width,
                            int output_height)
    : output_width(output_width), output_height(output_height),
      fb_size(output_width * output_height * 3 * sizeof(float)) {
  checkCudaError(hipMallocManaged((void **)&fb, fb_size));

  checkCudaError(hipMalloc((void **)&d_renderer, sizeof(DRenderer *)));

  setupRenderer<<<1, 1>>>(d_renderer, d_camera, output_width, output_height);

  checkCudaError(hipGetLastError());
  checkCudaError(hipDeviceSynchronize());
}

__global__ void setupRenderer(DRenderer **d_renderer, Camera **d_camera,
                              int output_width, int output_height) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    *d_renderer = new DRenderer(d_camera, output_width, output_height);

    (*d_renderer)->num_samples = 100;
    (*d_renderer)->num_bounces = 50;
  }
}

__device__ DRenderer::DRenderer(Camera **camera, int output_width,
                                int output_height)
    : _camera(camera), output_width(output_width), output_height(output_height),
      _fb_size(output_width * output_height) {
  setCamera(camera);
}

__device__ Camera **DRenderer::getCamera() const { return _camera; }

__device__ void DRenderer::setCamera(Camera **camera) {
  _camera = camera;

  auto viewport_u = (*camera)->getViewportU();
  auto viewport_v = (*camera)->getViewportV();

  _pixel_delta_u = viewport_u / output_width;
  _pixel_delta_v = viewport_v / output_height;

  _pixel00 = (*camera)->getViewportUpperLeft() +
             0.5 * (_pixel_delta_u + _pixel_delta_v);

  _center = (*camera)->getPosition();
}

__host__ void Renderer::render(Scene **scene, hiprandState *d_rand_state) {
  int NUM_THREADS_X = 8;
  int NUM_THREADS_Y = 8;

  dim3 blocks(output_width / NUM_THREADS_X + 1,
              output_height / NUM_THREADS_Y + 1);
  dim3 threads(NUM_THREADS_X, NUM_THREADS_Y);

  p_render<<<blocks, threads>>>(scene, d_renderer, this->fb, d_rand_state);

  checkCudaError(hipDeviceSynchronize());
}

__global__ void p_render(Scene **scene, DRenderer **d_renderer, float *fb,
                         hiprandState *rand_state) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i >= (*d_renderer)->output_width || j >= (*d_renderer)->output_height) {
    return;
  }

  int pixel_index = 3 * (i + j * (*d_renderer)->output_width);

  hiprand_init(2024, pixel_index / 3, 0, &rand_state[pixel_index / 3]);
  hiprandState local_rand_state = rand_state[pixel_index / 3];

  Color pixel_color(0, 0, 0);

  for (int sample = 0; sample < (*d_renderer)->num_samples; ++sample) {
    Ray ray = (*d_renderer)->getRay(i, j, &local_rand_state);
    pixel_color += (*d_renderer)
                       ->getRayColor(ray, scene, (*d_renderer)->num_bounces,
                                     &local_rand_state);
  }

  // Process samples

  pixel_color /= (*d_renderer)->num_samples;

  auto r = pixel_color.x;
  auto g = pixel_color.y;
  auto b = pixel_color.z;

  // Apply linear to gamma transform

  r = pow(r, 0.4545f);
  g = pow(g, 0.4545f);
  b = pow(b, 0.4545f);

  //  Write color

  fb[pixel_index] = r < 0.0 ? 0.0 : (r > 0.99 ? 0.99 : r);
  fb[pixel_index + 1] = g < 0.0 ? 0.0 : (g > 0.99 ? 0.99 : g);
  fb[pixel_index + 2] = b < 0.0 ? 0.0 : (b > 0.99 ? 0.99 : b);
}

// PRIVATE (?)

__device__ Ray DRenderer::getRay(int i, int j,
                                 hiprandState *local_rand_state) const {
  auto pixel_center = _pixel00 + i * _pixel_delta_u + j * _pixel_delta_v;
  auto pixel_sample = pixel_center + getPixelSampleSquare(local_rand_state);

  return Ray(_center, pixel_sample - _center);
}

__device__ Point3
DRenderer::getPixelSampleSquare(hiprandState *local_rand_state) const {
  auto x = -0.5 + random_float(local_rand_state);
  auto y = -0.5 + random_float(local_rand_state);

  return x * _pixel_delta_u + y * _pixel_delta_v;
}

__device__ Color DRenderer::getRayColor(const Ray &ray, Scene **scene,
                                        int num_bounces,
                                        hiprandState *local_rand_state) const {
  Ray r = ray;
  float attenuation = 1.0f;

  for (int i = 0; i < num_bounces; ++i) {
    HitRecord rec;

    if ((*scene)->hit(r, 0.001, INFINITY, rec)) {
      Vec3 direction = rec.normal + Vec3::randomUnit(local_rand_state);

      attenuation *= 0.5;
      r = Ray(rec.p, direction);
    } else {
      Vec3 unit_direction = r.direction.normalize();
      auto a = 0.5 * (unit_direction.y + 1.0);
      Color color = (1.0 - a) * Color(1, 1, 1) + a * Color(0.5, 0.7, 1.0);

      return color * attenuation;
    }
  }

  return Color();
}
