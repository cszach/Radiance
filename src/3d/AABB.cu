#include "hip/hip_runtime.h"
#include "AABB.cuh"
#include "Interval.cuh"
#include "Vec3.cuh"

__device__ AABB::AABB() {}
__device__ AABB::AABB(const Interval &_x, const Interval &_y,
                      const Interval &_z)
    : x(_x), y(_y), z(_z) {}
__device__ AABB::AABB(const Point3 &a, const Point3 &b)
    : x(Interval(fmin(a.x, b.x), fmax(a.x, b.x))),
      y(Interval(fmin(a.y, b.y), fmax(a.y, b.y))),
      z(Interval(fmin(a.z, b.z), fmax(a.z, b.z))) {}
__device__ AABB::AABB(const AABB &a, const AABB &b)
    : x(Interval(a.x, b.x)), y(Interval(a.y, b.y)), z(Interval(a.z, b.z)) {}

__device__ const Interval &AABB::axis(int i) const {
  if (i == 0)
    return x;
  if (i == 1)
    return y;
  /* else */ return z;
}

__device__ bool AABB::hit(const Ray &ray, Interval ray_t) const {
  for (int a = 0; a < 3; a++) {
    const Interval &ax = axis(a);
    float rayOrigin = ray.origin.get(a);
    float rayDirection = ray.direction.get(a);

    float t0 = fmin((ax.min - rayOrigin) / rayDirection,
                    (ax.max - rayOrigin) / rayDirection);
    float t1 = fmax((ax.min - rayOrigin) / rayDirection,
                    (ax.max - rayOrigin) / rayDirection);

    ray_t.min = fmax(t0, ray_t.min);
    ray_t.max = fmin(t1, ray_t.max);

    if (ray_t.max <= ray_t.min) {
      return false;
    }

    return true;
  }

  // for (int a = 0; a < 3; a++) {
  //   float inverseDirection = 1.0f / r.direction.get(a);
  //   float origin = r.origin.get(a);

  //   float t0 = (axis(a).min - origin) * inverseDirection;
  //   float t1 = (axis(a).max - origin) * inverseDirection;

  //   if (inverseDirection < 0) {
  //     float temp = t0;
  //     t0 = t1;
  //     t1 = temp;
  //   }

  //   if (t0 > ray_t.min)
  //     ray_t.min = t0;
  //   if (t1 < ray_t.max)
  //     ray_t.max = t1;

  //   if (ray_t.max <= ray_t.min) {
  //     return false;
  //   }
  // }

  // return true;
}