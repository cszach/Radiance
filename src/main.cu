#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <iostream>
#include <memory>
#include <stdio.h>

#include "Renderer.hpp"
#include "Scene.hpp"
#include "Sphere.hpp"
#include "cuda_helper.hpp"

__global__ void setup(Object **d_objects, Scene **d_scene, Camera **d_camera,
                      float vertical_fov, float aspect_ratio) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    auto test_sphere = new Sphere(0.5);
    test_sphere->setPosition(Point3(0, 0, -1));

    auto floor = new Sphere(100);
    floor->setPosition(Point3(0, -100.5, -1));

    *(d_objects) = test_sphere;
    *(d_objects + 1) = floor;

    *d_scene = new Scene(d_objects, 2);

    *d_camera = new Camera(vertical_fov, aspect_ratio);
  }
}

int main() {
  // Image

  const float ASPECT_RATIO = 2.0;
  const float VERTICAL_FOV = 100;
  const int IMAGE_WIDTH = 1200;
  const int IMAGE_HEIGHT = static_cast<int>(IMAGE_WIDTH / ASPECT_RATIO);
  const int NUM_OBJECTS = 2;

  std::ofstream f_out("image.ppm");

  // Camera

  Camera **d_camera;
  checkCudaError(hipMalloc((void **)&d_camera, sizeof(Camera *)));

  // Scene

  Object **d_objects;
  checkCudaError(
      hipMalloc((void **)&d_objects, NUM_OBJECTS * sizeof(Object *)));

  Scene **d_scene;
  checkCudaError(hipMalloc((void **)&d_scene, sizeof(Scene *)));

  // Render
  hiprandState *d_rand_state;
  checkCudaError(hipMalloc((void **)&d_rand_state,
                            IMAGE_WIDTH * IMAGE_HEIGHT * sizeof(hiprandState)));

  setup<<<1, 1>>>(d_objects, d_scene, d_camera, VERTICAL_FOV, ASPECT_RATIO);

  checkCudaError(hipGetLastError());
  checkCudaError(hipDeviceSynchronize());

  Renderer renderer(d_camera, IMAGE_WIDTH, IMAGE_HEIGHT);

  renderer.render(d_scene, d_rand_state);

  // Write to PPM

  f_out << "P3\n" << IMAGE_WIDTH << ' ' << IMAGE_HEIGHT << '\n' << 255 << '\n';

  for (int j = 0; j < IMAGE_HEIGHT; ++j) {
    for (int i = 0; i < IMAGE_WIDTH; ++i) {
      int pixel_index = 3 * (i + j * IMAGE_WIDTH);

      float r = renderer.fb[pixel_index + 0];
      float g = renderer.fb[pixel_index + 1];
      float b = renderer.fb[pixel_index + 2];

      auto ir = int(255.99 * r);
      auto ig = int(255.99 * g);
      auto ib = int(255.99 * b);

      f_out << ir << ' ' << ig << ' ' << ib << '\n';
    }
  }

  checkCudaError(hipDeviceReset());
}
